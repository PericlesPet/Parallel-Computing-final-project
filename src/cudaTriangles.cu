#include "hip/hip_runtime.h"
#include "cudaTriangles.h"


__global__ void triangleSum(int *rowIndex_dev, int *colIndex_dev, pair *pairs_cm_dev, pair *pairs_rm_dev, int nze, int N, int *triangle_sum){
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    int sum_i;


    // int minBlocks = ceilf((float) N / (float) blockDim.x);
    int minBlocks = ceilf((float) nze / (float) blockDim.x);
    
    // printf("tid = %d, i = %d, stride = %d, N = %d, minBlocks = %d, blockDim = %d, minBlocks*blockDim = %d \n", tid,i,stride, N, minBlocks, blockDim.x, minBlocks*blockDim.x);
    // if(i<nze){
    // if(i==0){
        //     printf("minBlocks = %d")
    // }

    for(int index=i; index<minBlocks*blockDim.x;index+=stride){
        
        if(tid ==0){
            // printf(" ---- tid = %d, i = %d, stride = %d, N = %d, rowIndex_dev[0] = %d \n", tid,i,stride, N, rowIndex_dev[0] );
        }
        
        // if(index==8){
            // printf("lololol \n");
            if(index<nze){
            // sum_i = 1;
            // sum_i = pairs_cm_dev[index].row;
            sum_i = sumForPair(rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N, index);
            // sum_i = rowIndex_dev[index];
        }else{
            sum_i = 0;
        }
        // printf("tid = %d, i = %d, stride = %d, nze = %d, minBlocks = %d, blockDim = %d, index = %d, sum_i = %d \n", tid,i,stride, nze, minBlocks, blockDim.x, index, sum_i);
        // }
        
        
        
        
        
        
        // map reduce the sums of each pair 
        // sdata[tid] = rowIndex_dev[index];
        // printf(" <<>> tid = %d, i = %d, stride = %d, N = %d \n", tid,i,stride, N );
        sdata[tid] = sum_i;
        __syncthreads();
        
        for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }
        // write result for this block to global mem
        if (tid == 0){
            triangle_sum[blockIdx.x] += sdata[0];
            // printf("TriangleSum[%d] = %d \n\n",blockIdx.x,triangle_sum[blockIdx.x]);
        }   
        
    }
        
}
    


//returns the final result of matrix A*A.*A for position (pair[index].row , pair[index].col)
__device__ int sumForPair(int *rowIndex_dev, int *colIndex_dev, pair *pairs_cm_dev, pair *pairs_rm_dev, int nze, int N, int index){
    int row = pairs_rm_dev[index].row-1;
    int col = pairs_rm_dev[index].col-1;
    // printf(" XyXyX -- row = %d , col = %d \n",row,col);
    int *row_arr;
    int *col_arr;
    // int *row_arr = row_arr_p[row];
    // int *col_arr = row_arr_p[col];
    
    int rowNzeCount = 0;// = row_arr[0];
    int colNzeCount = 0; //= col_arr[0];
    // int rowNzeCount = row_arr[0];
    // int colNzeCount = col_arr[0];

    // printf("rowNzeCount & col = %d, %d \n", rowNzeCount, colNzeCount);
    allRowNze(row, &row_arr, &rowNzeCount, rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N);
    allRowNze(col, &col_arr, &colNzeCount, rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N);

    int pairResult = commonElementCount(row_arr, rowNzeCount, col_arr,colNzeCount, row, col); 

    free(row_arr);
    free(col_arr);
    //  = (int*)malloc(sizeof(int)*10);
    // printf("<---> sum for pair (%d, %d) = %d \n", col,row,pairResult);
    return pairResult;
}

// assign to *row_arr matrix all non-zero-elements of A's "row" row.
__device__ void allRowNze(int row, int **row_arr,int *rowNzeCount, int *rowIndex_dev, int *colIndex_dev, pair *pairs_cm_dev, pair *pairs_rm_dev, int nze, int N){
    int colElems = colIndex_dev[row+1]-colIndex_dev[row];
    int rowElems = rowIndex_dev[row+1]-rowIndex_dev[row];
    
    // to avoid extreme situations for out of bounds behavior... 
    int kappa = N;
    if(row==kappa-1){
        // printf("")
        colElems = nze-colIndex_dev[row];
        rowElems = nze-rowIndex_dev[row];   
    }

    int staticCol = colElems;
    int staticrow = rowElems;

    // printf("row = %d, colElems = %d, rowElems = %d \n", row, colElems, rowElems);
    //total elements =  col elems + row elems
    (*row_arr) = (int *)malloc(sizeof(int)*(colElems+rowElems+1));
    (*row_arr)[0] = colElems + rowElems;
    (*rowNzeCount) = (*row_arr)[0];
    // need 2 pairs to calculate distance between them
    struct pair prevElem;
    
    prevElem.row = 1;
    prevElem.col = row;   // ok thats a little mindfuck but its correct
    
    struct pair nextElem;

    int count = 0;
    int dist = 0;
    int totalDist = 0;
    
    while(colElems>0){

        nextElem = pairs_cm_dev[colIndex_dev[row]+count];  // get from 'row'-th column the 'count'-th nz element
        dist = (nextElem.row - prevElem.row) + (nextElem.col - prevElem.col);
        totalDist += dist;
        (*row_arr)[count+1] = totalDist;
        
        count ++;
        prevElem = nextElem;
        colElems--;
    }
    
    while(rowElems>0){
        
        nextElem = pairs_rm_dev[rowIndex_dev[row] + count - staticCol];  // get from 'row'-th rowumn the 'count-colElems'-th nz element
        dist = (nextElem.row - prevElem.row) + (nextElem.col - prevElem.col);
        totalDist += dist;
        (*row_arr)[count+1] = totalDist;
        
        count ++;
        prevElem = nextElem;
        rowElems--;
    }
    
    // if(count == (colElems+rowElems)){
        // printf("- - - YES: row = %d, rowNzeCount = %d, colElems = %d, rowElems = %d , count = %d\n",row,(*rowNzeCount),(staticCol),(staticrow),count);
    // }else{
        // printf("^ ^ ^ NO: row = %d, rowNzeCount = %d, colElems = %d, rowElems = %d , count = %d\n",row,(*rowNzeCount),(staticCol),(staticrow),count);
        // printf("nooooo\n");
    // }
    
    
    // *row_arr = (int *)malloc(sizeof(int)*10);
    // (*row_arr)[i] = 5;
}


__device__ int commonElementCount(int *row_arr, int rowNzeCount, int *col_arr,int colNzeCount, int row, int col){
    
    int rowCount = 0;
    int colCount = 0;
    int commonElements = 0;
    int intex = threadIdx.x;

    while(rowCount<rowNzeCount && colCount<colNzeCount){

        if(row_arr[rowCount+1]==col_arr[colCount+1]){
            commonElements++;
            rowCount++;
            colCount++;
    
        }else if(row_arr[rowCount+1]>col_arr[colCount+1]){        
            colCount++;
        }else if(row_arr[rowCount+1]<col_arr[colCount+1]){
            rowCount++;
        }

    }
    // int rowCount = rowNzeCount;
    // int colCount = colNzeCount;

    printf(">>>Row %d : elems = %d [", row, row_arr[0]);

    for(int i=1;i<=rowNzeCount;i++){
        printf(" %d",row_arr[i]);
        // if(intex ==0){
        // }
    }    
    printf("\n");
    // printf(" ]\n");

    printf(">>>Col %d : elems = %d [", col, col_arr[0]);
    for(int i=1;i<=colNzeCount;i++){
        printf("%d ",col_arr[i]);
    }    
    printf("\n");
    printf(">>> (%d X %d) common: %d \n", col+1, row+1, commonElements );

    // printf("")
    return commonElements;
}