#include "hip/hip_runtime.h"
#include <cudaTriangles.h>


__global__ void triangleSum(int *rowIndex_dev, int *colIndex_dev, pair *pairs_cm_dev, pair *pairs_rm_dev, int nze, int N, int *triangle_sum){
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    int sum_i;

    int minBlocks = ceilf((float) N / (float) blockDim.x);
    // int minBlocks = ceilf((float) nze / (float) blockDim.x);
    
    // printf("tid = %d, i = %d, stride = %d, N = %d, minBlocks = %d, blockDim = %d, minBlocks*blockDim = %d \n", tid,i,stride, N, minBlocks, blockDim.x, minBlocks*blockDim.x);
    // if(i<nze){
    // if(i==0){
        //     printf("minBlocks = %d")
    // }

    for(int index=i; index<minBlocks*blockDim.x;index+=stride){
        if(tid ==0){
            printf(" ---- tid = %d, i = %d, stride = %d, N = %d, rowIndex_dev[0] = %d \n", tid,i,stride, N, rowIndex_dev[0] );
        }
        
        if(index<N){
            sum_i = rowIndex_dev[index];
        }else{
            sum_i = 0;
        }
        printf("tid = %d, i = %d, stride = %d, N = %d, minBlocks = %d, blockDim = %d, index = %d, sum_i = %d \n", tid,i,stride, N, minBlocks, blockDim.x, index, sum_i);
        // }
        
        
        
        
        
        
        // map reduce the sums of each pair 
        // sdata[tid] = rowIndex_dev[index];
        // printf(" <<>> tid = %d, i = %d, stride = %d, N = %d \n", tid,i,stride, N );
        sdata[tid] = sum_i;
        __syncthreads();
        
        for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }
        // write result for this block to global mem
        if (tid == 0){
            triangle_sum[blockIdx.x] += sdata[0];
            printf("TriangleSum[%d] = %d \n\n",blockIdx.x,triangle_sum[blockIdx.x]);
        }   
        
    }
        
}
    