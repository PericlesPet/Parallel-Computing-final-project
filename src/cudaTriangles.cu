#include "hip/hip_runtime.h"
#include <cudaTriangles.h>


template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
// __global__ void reduce6(int *g_idata, int *g_odata, unsigned int n)
__global__ void triangleSum(int *rowIndex_dev, int *colIndex_dev, pair *pairs_cm_dev, pair *pairs_rm_dev, int nze, int N, int *triangle_sum){
    extern __shared__ sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;






    // map reduce the sums of each pair 
    sdata[tid] = sum_i;
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem

    if (tid == 0) (*triangle_sum) = sdata[0];


}
