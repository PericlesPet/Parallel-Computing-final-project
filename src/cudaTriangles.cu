#include "hip/hip_runtime.h"
#include <cudaTriangles.h>


__global__ void triangleSum(int *rowIndex_dev, int *colIndex_dev, pair *pairs_cm_dev, pair *pairs_rm_dev, int nze, int N, int *triangle_sum){
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    int sum_i;

    // int minBlocks = ceilf((float) N / (float) blockDim.x);
    int minBlocks = ceilf((float) nze / (float) blockDim.x);
    
    // printf("tid = %d, i = %d, stride = %d, N = %d, minBlocks = %d, blockDim = %d, minBlocks*blockDim = %d \n", tid,i,stride, N, minBlocks, blockDim.x, minBlocks*blockDim.x);
    // if(i<nze){
    // if(i==0){
        //     printf("minBlocks = %d")
    // }

    for(int index=i; index<minBlocks*blockDim.x;index+=stride){
        if(tid ==0){
            printf(" ---- tid = %d, i = %d, stride = %d, N = %d, rowIndex_dev[0] = %d \n", tid,i,stride, N, rowIndex_dev[0] );
        }
        
        if(index<nze){
            // sum_i = pairs_cm_dev[index].row;
            sum_i = sumForPair(rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N, index);
            // sum_i = rowIndex_dev[index];
        }else{
            sum_i = 0;
        }
        // printf("tid = %d, i = %d, stride = %d, nze = %d, minBlocks = %d, blockDim = %d, index = %d, sum_i = %d \n", tid,i,stride, nze, minBlocks, blockDim.x, index, sum_i);
        // }
        
        
        
        
        
        
        // map reduce the sums of each pair 
        // sdata[tid] = rowIndex_dev[index];
        // printf(" <<>> tid = %d, i = %d, stride = %d, N = %d \n", tid,i,stride, N );
        sdata[tid] = sum_i;
        __syncthreads();
        
        for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }
        // write result for this block to global mem
        if (tid == 0){
            triangle_sum[blockIdx.x] += sdata[0];
            printf("TriangleSum[%d] = %d \n\n",blockIdx.x,triangle_sum[blockIdx.x]);
        }   
        
    }
        
}
    


//returns the final result of matrix A*A.*A for position (pair[index].row , pair[index].col)
__device__ int sumForPair(int *rowIndex_dev, int *colIndex_dev, pair *pairs_cm_dev, pair *pairs_rm_dev, int nze, int N, int index){
    int row = pairs_cm_dev[index].row;
    int col = pairs_cm_dev[index].col;
    int *row_arr;
    int *col_arr;
    
    int rowNzeCount = 0;
    int colNzeCount = 0;

    allRowNze(row, &row_arr, &rowNzeCount, rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N);
    allRowNze(col, &col_arr, &colNzeCount, rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N);

    int pairResult = commonElementCount(row_arr, rowNzeCount, col_arr,colNzeCount); 

    free(row_arr);
    free(col_arr);
    //  = (int*)malloc(sizeof(int)*10);
    return pairResult;
}

// assign to *row_arr matrix all non-zero-elements of A's "row" row.
__device__ void allRowNze(int row, int **row_arr,int *rowNzeCount, int *rowIndex_dev, int *colIndex_dev, pair *pairs_cm_dev, pair *pairs_rm_dev, int nze, int N){

    *row_arr = (int *)malloc(sizeof(int)*10);
    // (*row_arr)[i] = 5;
}

__device__ int commonElementCount(int *row_arr, int rowNzeCount, int *col_arr,int colNzeCount){
    return 1;
}