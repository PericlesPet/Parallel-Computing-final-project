#include "hip/hip_runtime.h"

#include "test.h"
// #include "args.h"
#include "utils.h"
#include "cudaTriangles.h"

#include <unistd.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds. 
// reference: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/coalescing-global/coalescing.cu
// e.g.: checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );
// e.g.: kernel<<< x,y >>>()
//       checkCuda( hipGetLastError() );
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || !defined(DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


int main(void)
{
  
  // PARAMETERS
  int blockMultiplier = 32;
  int threadMultiplier = 1;


  //VAR DECLARATIONS
  int *rowVec;
  int *colVec;
  int N;
  int nze;
  int *rowIndex;
  int *colIndex;
  // char *filepath = "graphs/chesapeake.mtx";
  char *filepath = "graphs/auto.mtx";
  // char *filepath = "graphs/delaunay_n10.mtx";
  
  
  double time_start, time_end;

  printf("preprocessing...\n");
  time_start = get_time();
  // READ SPARSE MATRIX FROM FILE
  readMtxFile(filepath, &rowVec, &colVec, &N, &nze);
  
  //  STORE ROWS  : from rowVec, colVec --> pairs_rm, rowIndex
  
  // Find indeces of separate sparse rows --> assigns rowIndex array
  separateRows(nze, N, rowVec, colVec, &rowIndex);
  
  //printf for each (row, col):
  
  //row major pair array
  struct pair *pairs_rm;
  struct pair *pairs_rm_dev;
  hipHostMalloc(&pairs_rm,sizeof(pair)*nze);
  hipMalloc(&pairs_rm_dev,sizeof(pair)*nze);
  // unify vectors into pair array
  arraysToPairs(rowVec, colVec, nze, pairs_rm);
  
  // for(int i=0;i<nze;i++){
  //     printf("%d. (col,row) = (%d, %d) \n",i, pairs_rm[i].col, pairs_rm[i].row);
  // }
    // for(int i=0;i<nze;i++){
    //     printf("%d. (col,row) = (%d, %d) , pair = (%d, %d) -- arr_rm index = [%d + count]\n",i, colVec[i], rowVec[i], pairs_rm[i].col, pairs_rm[i].row, rowIndex[rowVec[i]-1]);  
    // }  
    // printf("\n");
      printf("nze = %d, colVec[nze] = %d, colVec[nze] = %d\n", nze, colVec[nze-1], rowVec[nze-1]);
      // Sort vectors Column-wise

      pairsort(colVec, rowVec, nze);
      printf("\n\n");
      // COLUMNS
      // Find indeces of separate sparse columns --> assigns colIndex array
      separateRows(nze,N, colVec, rowVec, &colIndex);
      //column major pair array
      struct pair *pairs_cm;
      struct pair *pairs_cm_dev;
      hipHostMalloc(&pairs_cm,sizeof(pair)*nze);
      hipMalloc(&pairs_cm_dev, sizeof(pair)*nze);
      // unify vectors into pair array
      arraysToPairs(rowVec, colVec, nze, pairs_cm);
      
// for(int i=0;i<nze;i++){
//     printf("%d. (col,row) = (%d, %d) -- col_cm index = [%d + count] \n",i, colVec[i], rowVec[i], colIndex[colVec[i]-1]);
  
//   }  
  
// struct pair *pairs_cm_dev, *pairs_rm_dev;
int *colIndex_dev, *rowIndex_dev;

hipMalloc(&colIndex_dev, sizeof(int)*N);
hipMalloc(&rowIndex_dev, sizeof(int)*N);

// declare pair arrays directly for device use
hipMemcpy(pairs_cm_dev,pairs_cm, sizeof(pair)*nze,hipMemcpyHostToDevice);
hipMemcpy(pairs_rm_dev,pairs_rm, sizeof(pair)*nze,hipMemcpyHostToDevice);
hipMemcpy(colIndex_dev,colIndex, sizeof(int)*N,hipMemcpyHostToDevice);
hipMemcpy(rowIndex_dev,rowIndex, sizeof(int)*N,hipMemcpyHostToDevice);

// colVec & rowVec no longer needed
free(colVec);
free(rowVec);

// for(int i=0;i<nze;i++){
//   printf("%d. pair = (%d, %d) \n",i, pairs_rm[i].col, pairs_rm[i].row);
  
// }  
  // Get Device Properties 
  // printf("checking cuda\n");
  int deviceId;
  checkCuda(hipGetDevice(&deviceId));
  hipDeviceProp_t props;
  checkCuda(hipGetDeviceProperties(&props, deviceId));
  int warpsize = props.warpSize;         // Warp Size
  int SMs = props.multiProcessorCount;  //Streaming Multiprocessors

  int blocks = blockMultiplier * SMs;
  int threads = threadMultiplier * warpsize; 

  // printf("blocks = %d, threads = %d \n",blocks,threads);
  
  // triangleSum array will have ceil(nze/blockDim.x) / blocks size
  int *triangleSum_host;
  hipHostMalloc(&triangleSum_host, sizeof(int)*blocks);
  int *triangleSum_dev;
  hipMalloc(&triangleSum_dev,sizeof(int)*blocks);
  
  
  
  // create pointer to pointer for rows -> row[i][n] : i'th row, (n-1)'th element
  
  
  
  
  int **row_arr_pointer_device;
  int **row_arr_pointer_host = (int **)malloc(sizeof(int*)*N);
  
  checkCuda(hipMalloc(&row_arr_pointer_device, sizeof(int)*N));

  
  int *row_arr_dev;
  // , *row_arr_host;
  
  int rowNzeCount = 0;
  
  // hipHostMalloc(&row_arr_pointer, sizeof(int)*N);
  
  // hipHostMalloc(&row_arr_host, )
  int nzeCummu = 0;
  int *nzeCummus = (int *)malloc(sizeof(int)*N);
  int *allRowsArray = (int *)malloc(sizeof(int)*(2*nze+N));
  
  for(int i=0; i<N;i++){
    // printf("i = %d \n\n",i);
    // allRowNze(i, &row_arr_pointer[i],&row_arr_pointer[i][0], rowIndex, colIndex, pairs_cm, pairs_rm, nze, N);
    allRowNze(i, &(row_arr_pointer_host[i]),&rowNzeCount, rowIndex, colIndex, pairs_cm, pairs_rm, nze, N);
    // allRowNze(i,&(allRowsArray)+nzeCummu,&rowNzeCount, rowIndex, colIndex, pairs_cm, pairs_rm, nze, N);
    // allRowNze(i,&allRowsArray[nzeCummu],&rowNzeCount, rowIndex, colIndex, pairs_cm, pairs_rm, nze, N);
    
    hipMalloc(&row_arr_dev, sizeof(int)*rowNzeCount);
    
    nzeCummus[i] = nzeCummu;
    
    // &(allRowsArray+nzeCummu) 
    
    nzeCummu += rowNzeCount+1;
    
    allRowsArray[nzeCummus[i]] = row_arr_pointer_host[i][0];
    // printf("Row %d : %d [", i, allRowsArray[nzeCummus[i]]);
    
    for(int j= 1 ;j<=rowNzeCount;j++){
      // printf(" %d" ,row_arr_pointer_host[i][j]);
      allRowsArray[j + nzeCummus[i]] = row_arr_pointer_host[i][j];
      // printf( " %d" ,allRowsArray[ j + nzeCummus[i] ] );
    }
    // printf(" ]\n");
    
    
    
    // // hipMemcpy(&row_arr_dev, &row_arr_pointer_host[i], sizeof(int)*rowNzeCount, hipMemcpyHostToDevice);
    
    // hipMemcpy(&row_arr_pointer_device[i], &row_arr_pointer_host[i], sizeof(int)*rowNzeCount, hipMemcpyHostToDevice);
    // checkCuda(hipMalloc(&row_arr_pointer_device[i], rowNzeCount*sizeof(int)));
    // checkCuda(hipMemcpy(row_arr_pointer_device[i], row_arr_pointer_host[i], sizeof(int)*rowNzeCount, hipMemcpyHostToDevice));
    
    // // hipMalloc(&(row_arr_pointer_device[i]), sizeof(int)*rowNzeCount);
    // // hipMemcpy(&row_arr_pointer_device[i],&row_arr_host,sizeof(int) )
  }  

  int *nzeCummus_dev;
  checkCuda(hipMalloc(&nzeCummus_dev, sizeof(int)*N));
  int *allRowsArray_dev;
  checkCuda(hipMalloc(&allRowsArray_dev, sizeof(int)*(2*nze+N)));

  
  checkCuda(hipMemcpy(nzeCummus_dev, nzeCummus, sizeof(int)*N, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(allRowsArray_dev, allRowsArray, sizeof(int)*(2*nze+N), hipMemcpyHostToDevice));
  

  
  


  
  // printf("time_start 2 = %f \n", time_start);
  
  time_end = get_time();
  printf("preprocessing took: %f secs \n", time_end-time_start);
  
  printf("\ninitiating kernel with: ");
  printf("blocks = %d, threads = %d \n",blocks,threads);
  time_start = get_time();

  // triangleSum<<<blocks,threads,sizeof(int)*threads>>>(rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N, triangleSum_dev);
  // triangleSum<<<blocks,threads,sizeof(int)*threads>>>(row_arr_pointer_device, pairs_rm_dev, nze, N, triangleSum_dev);
  triangleSum<<<blocks,threads,sizeof(int)*threads>>>(allRowsArray_dev, nzeCummus_dev, pairs_rm_dev, nze, N, triangleSum_dev);
  
  checkCuda( hipGetLastError() );
  checkCuda(hipDeviceSynchronize());
  time_end = get_time();
  // printf("time_end = %f \n", time_end);
  checkCuda(hipMemcpy(triangleSum_host, triangleSum_dev,sizeof(int)*blocks,hipMemcpyDeviceToHost));  
  checkCuda(hipFree(triangleSum_dev));

  // printf(" --> sum is: \n");
  // for(int i=0;i<blocks;i++){
  //   printf(" ooo array = %d\n",triangleSum_host[i]);
  // }
  int cudaSum = quickSum(triangleSum_host, blocks);

  int *quickArr = (int *)malloc(sizeof(int)*nze);
  
  for(int i=0;i<nze;i++){
    quickArr[i] = pairs_cm[i].row;
  }
  // int realSum = quickSum(quickArr, nze);
  // int realSum = quickSum(rowIndex, N);
  printf("--> Result is: %d\n \n",cudaSum/3);
  printf(" time: %f\n",time_end-time_start);
  // for(int i=0;i<N;i++){
  //   printf("%d. (%d) \n",i,rowIndex[i]);
  //   // printf("%d. (%d , %d) \n\n",i,pairs_cm[i].col,pairs_cm[i].row);
  // }
  



  // hipFree(x);
  // hipFree(y);
  
  return 0;
}


