#include "hip/hip_runtime.h"

#include "test.h"
// #include "args.h"
#include "utils.h"
#include "cudaTriangles.h"

#include <unistd.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds. 
// reference: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/coalescing-global/coalescing.cu
// e.g.: checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );
// e.g.: kernel<<< x,y >>>()
//       checkCuda( hipGetLastError() );
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || !defined(DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  #endif
  return result;
}


int main(void)
{
  
  // PARAMETERS
  int blockMultiplier = 32;
  int threadMultiplier = 2;
  
  // char *filepath = "graphs/chesapeake.mtx";
  char *filepath = "graphs/auto.mtx";
  // char *filepath = "graphs/great-britain_osm.mtx";
  // char *filepath = "graphs/delaunay_n22.mtx";
  // char *filepath = "graphs/delaunay_n10.mtx";

  //VAR DECLARATIONS
  int *rowVec;
  int *colVec;
  int N;
  int nze;
  int *rowIndex;
  int *colIndex;
  
  double time_start, time_end;

  printf("preprocessing...\n");
  time_start = get_time();
  // READ SPARSE MATRIX FROM FILE
  readMtxFile(filepath, &rowVec, &colVec, &N, &nze);
  
  //  STORE ROWS  : from rowVec, colVec --> pairs_rm, rowIndex
  
  // Find indeces of separate sparse rows --> assigns rowIndex array
  separateRows(nze, N, rowVec, colVec, &rowIndex);
  
  //row major pair array
  struct pair *pairs_rm;
  struct pair *pairs_rm_dev;
  hipHostMalloc(&pairs_rm,sizeof(pair)*nze);
  hipMalloc(&pairs_rm_dev,sizeof(pair)*nze);
  // unify vectors into pair array
  arraysToPairs(rowVec, colVec, nze, pairs_rm);
  
  printf("nze = %d, colVec[nze] = %d, colVec[nze] = %d\n", nze, colVec[nze-1], rowVec[nze-1]);

  // Sort vectors Column-wise
  pairsort(colVec, rowVec, nze);
  printf("\n\n");
  // COLUMNS
  // Find indeces of separate sparse columns --> assigns colIndex array
  separateRows(nze,N, colVec, rowVec, &colIndex);
  //column major pair array
  struct pair *pairs_cm;
  struct pair *pairs_cm_dev;
  hipHostMalloc(&pairs_cm,sizeof(pair)*nze);
  hipMalloc(&pairs_cm_dev, sizeof(pair)*nze);
  // unify vectors into pair array
  arraysToPairs(rowVec, colVec, nze, pairs_cm);

  int *colIndex_dev, *rowIndex_dev;

  hipMalloc(&colIndex_dev, sizeof(int)*N);
  hipMalloc(&rowIndex_dev, sizeof(int)*N);

  // declare pair arrays directly for device use
  hipMemcpy(pairs_cm_dev,pairs_cm, sizeof(pair)*nze,hipMemcpyHostToDevice);
  hipMemcpy(pairs_rm_dev,pairs_rm, sizeof(pair)*nze,hipMemcpyHostToDevice);
  hipMemcpy(colIndex_dev,colIndex, sizeof(int)*N,hipMemcpyHostToDevice);
  hipMemcpy(rowIndex_dev,rowIndex, sizeof(int)*N,hipMemcpyHostToDevice);

  // colVec & rowVec no longer needed
  free(colVec);
  free(rowVec);


  // Get Device Properties 
  int deviceId;
  checkCuda(hipGetDevice(&deviceId));
  hipDeviceProp_t props;
  checkCuda(hipGetDeviceProperties(&props, deviceId));
  int warpsize = props.warpSize;         // Warp Size
  int SMs = props.multiProcessorCount;  //Streaming Multiprocessors

  int blocks = blockMultiplier * SMs;
  int threads = threadMultiplier * warpsize; 

  
  // triangleSum array will have ceil(nze/blockDim.x) / blocks size
  int *triangleSum_host;
  hipHostMalloc(&triangleSum_host, sizeof(int)*blocks);
  int *triangleSum_dev;
  hipMalloc(&triangleSum_dev,sizeof(int)*blocks);
  
  
  int **row_arr_pointer_device;
  int **row_arr_pointer_host = (int **)malloc(sizeof(int*)*N);
  
  checkCuda(hipMalloc(&row_arr_pointer_device, sizeof(int)*N));

  
  int *row_arr_dev;
  
  int rowNzeCount = 0;
  
  int nzeCummu = 0;
  int *nzeCummus = (int *)malloc(sizeof(int)*N);
  int *allRowsArray = (int *)malloc(sizeof(int)*(2*nze+N));
  
  for(int i=0; i<N;i++){
    // if(i%100000==0){
    //   printf("i = %d\n", i);
    // }
    allRowNze(i, &(row_arr_pointer_host[i]),&rowNzeCount, rowIndex, colIndex, pairs_cm, pairs_rm, nze, N);
    
    // hipMalloc(&row_arr_dev, sizeof(int)*rowNzeCount);
    
    nzeCummus[i] = nzeCummu;
    nzeCummu += rowNzeCount+1;
    
    allRowsArray[nzeCummus[i]] = row_arr_pointer_host[i][0];
    
    for(int j= 1 ;j<=rowNzeCount;j++){
      // printf(" %d" ,row_arr_pointer_host[i][j]);
      allRowsArray[j + nzeCummus[i]] = row_arr_pointer_host[i][j];
    }
  }  

  int *nzeCummus_dev;
  checkCuda(hipMalloc(&nzeCummus_dev, sizeof(int)*N));
  int *allRowsArray_dev;
  checkCuda(hipMalloc(&allRowsArray_dev, sizeof(int)*(2*nze+N)));

  
  checkCuda(hipMemcpy(nzeCummus_dev, nzeCummus, sizeof(int)*N, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(allRowsArray_dev, allRowsArray, sizeof(int)*(2*nze+N), hipMemcpyHostToDevice));
  

  
  


  
  // printf("time_start 2 = %f \n", time_start);
  
  time_end = get_time();
  printf("preprocessing took: %f secs \n", time_end-time_start);
  
  printf("\ninitiating kernel with: ");
  printf("blocks = %d, threads = %d \n",blocks,threads);
  time_start = get_time();

  // triangleSum<<<blocks,threads,sizeof(int)*threads>>>(rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N, triangleSum_dev);
  // triangleSum<<<blocks,threads,sizeof(int)*threads>>>(row_arr_pointer_device, pairs_rm_dev, nze, N, triangleSum_dev);
  triangleSum<<<blocks,threads,sizeof(int)*threads>>>(allRowsArray_dev, nzeCummus_dev, pairs_rm_dev, nze, N, triangleSum_dev);
  
  checkCuda( hipGetLastError() );
  checkCuda(hipDeviceSynchronize());
  time_end = get_time();
  // printf("time_end = %f \n", time_end);
  checkCuda(hipMemcpy(triangleSum_host, triangleSum_dev,sizeof(int)*blocks,hipMemcpyDeviceToHost));  
  checkCuda(hipFree(triangleSum_dev));

  // printf(" --> sum is: \n");
  // for(int i=0;i<blocks;i++){
  //   printf(" ooo array = %d\n",triangleSum_host[i]);
  // }
  int cudaSum = quickSum(triangleSum_host, blocks);

  int *quickArr = (int *)malloc(sizeof(int)*nze);
  
  for(int i=0;i<nze;i++){
    quickArr[i] = pairs_cm[i].row;
  }
  // int realSum = quickSum(quickArr, nze);
  // int realSum = quickSum(rowIndex, N);
  printf("--> Result is: %d\n \n",cudaSum/3);
  printf(" time: %f seconds\n",time_end-time_start);
  // for(int i=0;i<N;i++){
  //   printf("%d. (%d) \n",i,rowIndex[i]);
  //   // printf("%d. (%d , %d) \n\n",i,pairs_cm[i].col,pairs_cm[i].row);
  // }
  



  // hipFree(x);
  // hipFree(y);
  
  return 0;
}


