#include "hip/hip_runtime.h"

#include "test.h"
#include "args.h"
#include "utils.h"
#include "cudaTriangles.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds. 
// reference: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/coalescing-global/coalescing.cu
// e.g.: checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );
// e.g.: kernel<<< x,y >>>()
//       checkCuda( hipGetLastError() );
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


int main(void)
{
  
  // PARAMETERS
  int blockMultiplier = 1;
  int threadMultiplier = 1;


  //VAR DECLARATIONS
  int *rowVec;
  int *colVec;
  int N;
  int nze;
  int *rowIndex;
  int *colIndex;
  char *filepath = "graphs/chesapeake.mtx";
  

  // READ SPARSE MATRIX FROM FILE
  readMtxFile(filepath, &rowVec, &colVec, &N, &nze);
    
  //  STORE ROWS  : from rowVec, colVec --> pairs_rm, rowIndex

  // Find indeces of separate sparse rows --> assigns rowIndex array
  separateRows(nze, N, rowVec, colVec, &rowIndex);
  //row major pair array
  struct pair *pairs_rm;
  struct pair *pairs_rm_dev;
  hipHostMalloc(&pairs_rm,sizeof(pair)*nze);
  hipMalloc(&pairs_rm_dev,sizeof(pair)*nze);
  // unify vectors into pair array
  arraysToPairs(rowVec, colVec, nze, pairs_rm);
  
  // Sort vectors Column-wise
  pairsort(colVec, rowVec, nze);
  
  // COLUMNS
  // Find indeces of separate sparse columns --> assigns colIndex array
  separateRows(nze,N, colVec, rowVec, &colIndex);
  //column major pair array
  struct pair *pairs_cm;
  struct pair *pairs_cm_dev;
  hipHostMalloc(&pairs_cm,sizeof(pair)*nze);
  hipMalloc(&pairs_cm_dev, sizeof(pair)*nze);
  // unify vectors into pair array
  arraysToPairs(rowVec, colVec, nze, pairs_cm);


  // struct pair *pairs_cm_dev, *pairs_rm_dev;
  int *colIndex_dev, *rowIndex_dev;

  hipMalloc(&colIndex_dev, sizeof(int)*N);
  hipMalloc(&rowIndex_dev, sizeof(int)*N);

  // declare pair arrays directly for device use
  hipMemcpy(pairs_cm_dev,pairs_cm, sizeof(pair)*nze,hipMemcpyHostToDevice);
  hipMemcpy(pairs_rm_dev,pairs_rm, sizeof(pair)*nze,hipMemcpyHostToDevice);
  hipMemcpy(colIndex_dev,colIndex, sizeof(int)*N,hipMemcpyHostToDevice);
  hipMemcpy(rowIndex_dev,rowIndex, sizeof(int)*N,hipMemcpyHostToDevice);

  // colVec & rowVec no longer needed
  free(colVec);
  free(rowVec);
  
  // Get Device Properties 
  int deviceId;
  hipGetDevice(&deviceId);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  int warpsize = props.warpSize;         // Warp Size
  int SMs = props.multiProcessorCount;  //Streaming Multiprocessors

  int blocks = blockMultiplier * SMs;
  int threads = threadMultiplier * warpsize; 

  printf("blocks = %d, threads = %d \n",blocks,threads);
  
  // triangleSum array will have ceil(nze/blockDim.x) / blocks size
  int *triangleSum_host;
  hipHostMalloc(&triangleSum_host, sizeof(int)*blocks);
  int *triangleSum_dev;
  hipMalloc(&triangleSum_dev,sizeof(int)*blocks);


  triangleSum<<<blocks,threads,sizeof(int)*threads>>>(rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N, triangleSum_dev);

  checkCuda( hipGetLastError() );
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(triangleSum_host, triangleSum_dev,sizeof(int)*blocks,hipMemcpyDeviceToHost));  
  checkCuda(hipFree(triangleSum_dev));

  // printf(" --> sum is: \n");
  for(int i=0;i<blocks;i++){
    printf(" ooo array = %d\n",triangleSum_host[i]);
  }
  int cudaSum = quickSum(triangleSum_host, blocks);

  int *quickArr = (int *)malloc(sizeof(int)*nze);
  for(int i=0;i<nze;i++){
    quickArr[i] = pairs_cm[i].row;
  }
  int realSum = quickSum(quickArr, nze);
  // int realSum = quickSum(rowIndex, N);
  printf(" --> sum is: %d , realSum is: %d\n",cudaSum,realSum);
  // for(int i=0;i<N;i++){
  //   printf("%d. (%d) \n",i,rowIndex[i]);
  //   // printf("%d. (%d , %d) \n\n",i,pairs_cm[i].col,pairs_cm[i].row);
  // }
  



  // hipFree(x);
  // hipFree(y);
  
  return 0;
}


