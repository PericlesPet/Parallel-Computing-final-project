
#include "test.h"
#include "args.h"
#include "cudaTriangles.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
int main(int argc, char **argv)
{
    if (argc < 2) {
        fprintf(stderr, "usage: %s [test|]\n", argv[0]);
    } else if (0 == strcmp(argv[1], "test")){
        test();
    } else {
        fprintf(stderr, "unrecognized option: %s\n", argv[1]);
    }
    
    int sum=0;
    for (size_t i = 0; i < 5; i++)
    {
        sum+= i;
    }
    return 0;

}

