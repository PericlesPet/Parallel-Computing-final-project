#include "hip/hip_runtime.h"

#include "test.h"
#include "args.h"
#include "utils.h"
#include "cudaTriangles.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds. 
// reference: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/coalescing-global/coalescing.cu
// e.g.: checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );
// e.g.: kernel<<< x,y >>>()
//       checkCuda( hipGetLastError() );
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}



// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  
  // PARAMETERS
  int blockMultiplier = 4;
  int threadMultiplier = 4;


  //VAR DECLARATIONS
  int *rowVec;
  int *colVec;
  int N;
  int nze;
  int *rowIndex;
  int *colIndex;
  char *filepath = "graphs/chesapeake.mtx";
  
  readMtxFile(filepath, &rowVec, &colVec, &N, &nze);
  printf("hi");
    
  // ROWS
  // Find indeces of separate sparse rows --> assigns rowIndex array
  separateRows(nze, N, rowVec, colVec, &rowIndex);
  //row major pair array
  struct pair *pairs_rm;
  hipHostMalloc(&pairs_rm,sizeof(pair)*nze);
  // unify vectors into pair array
  arraysToPairs(rowVec, colVec, nze, pairs_rm);
  
  // Sort vectors Column-wise
  pairsort(colVec, rowVec, nze);
  
  // COLUMNS
  // Find indeces of separate sparse columns --> assigns colIndex array
  separateRows(nze,N, colVec, rowVec, &colIndex);
  //column major pair array
  struct pair *pairs_cm;
  hipHostMalloc(&pairs_cm,sizeof(pair)*nze);
  // unify vectors into pair array
  arraysToPairs(rowVec, colVec, nze, pairs_cm);


  struct pair *pairs_cm_dev, *pairs_rm_dev;
  int *colIndex_dev, *rowIndex_dev;

  // declare pair arrays directly for device use
  hipMemcpy(pairs_cm_dev,pairs_cm, sizeof(pair)*nze,hipMemcpyHostToDevice);
  hipMemcpy(pairs_rm_dev,pairs_rm, sizeof(pair)*nze,hipMemcpyHostToDevice);
  hipMemcpy(colIndex_dev,colIndex, sizeof(int)*nze,hipMemcpyHostToDevice);
  hipMemcpy(rowIndex_dev,rowIndex, sizeof(int)*nze,hipMemcpyHostToDevice);

  // colVec & rowVec no longer needed
  free(colVec);
  free(rowVec);
  
  // Get Device Properties 

  int deviceId;
  hipGetDevice(&deviceId);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  int warpsize = props.warpSize;         // Warp Size
  int SMs = props.multiProcessorCount;  //Streaming Multiprocessors

  int blocks = blockMultiplier * SMs;
  int threads = threadMultiplier * warpsize; 

  // triangle-sum<<<blocks,threads>>>(rowIndex_dev, colIndex_dev, pairs_cm_dev, pairs_rm_dev, nze, N);

  // for(int i=0;i<nze;i++){
  //   printf("%d. (%d , %d) \n",i,colVec[i],rowVec[i]);
  //   printf("%d. (%d , %d) \n\n",i,pairs_cm[i].col,pairs_cm[i].row);
  // }
  



  // // Allocate Unified Memory – accessible from CPU or GPU
  // hipMallocManaged(&x, N*sizeof(float));
  // hipMallocManaged(&y, N*sizeof(float));

  // // initialize x and y arrays on the host
  // for (int i = 0; i < N; i++) {
  //   x[i] = 1.0f;
  //   y[i] = 2.0f;
  // }

  // // Run kernel on 1M elements on the GPU
  // add<<<1, 1>>>(N, x, y);

  // // Wait for GPU to finish before accessing on host
  // hipDeviceSynchronize();

  // // Check for errors (all values should be 3.0f)
  // float maxError = 0.0f;
  // for (int i = 0; i < N; i++)
  //   maxError = fmax(maxError, fabs(y[i]-3.0f));
  // std::cout << "Max error: " << maxError << std::endl;

  // // Free memory
  // hipFree(x);
  // hipFree(y);
  
  return 0;
}


