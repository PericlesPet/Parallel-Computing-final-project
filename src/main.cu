#include "hip/hip_runtime.h"

#include "test.h"
#include "args.h"
#include "utils.h"
#include "cudaTriangles.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds. 
// reference: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/coalescing-global/coalescing.cu
// e.g.: checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );
// e.g.: kernel<<< x,y >>>()
//       checkCuda( hipGetLastError() );
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}



// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  // int N = 1<<20;
  // float *x, *y;

  int *rowVec;
  int *colVec;
  int N;
  int nze;

  int *rowIndex;
  int *colIndex;

  char *filepath = "graphs/chesapeake.mtx";
  
  readMtxFile(filepath, &rowVec, &colVec, &N, &nze);
  separateRows(nze, N, rowVec, colVec, &rowIndex);
  // printf("main: nze = %d, N = %d \n rowVec[0] = %d, colVec[0] = %d\n",nze,N,rowVec[0],colVec[0]);

  pairsort(colVec, rowVec, nze);
  
  separateRows(nze,N, colVec, rowVec, &colIndex);

  // for(int i=0;i<nze;i++){
  //   printf("%d. (%d , %d) \n",i,colVec[i],rowVec[i]);
  // }

  // // Allocate Unified Memory – accessible from CPU or GPU
  // hipMallocManaged(&x, N*sizeof(float));
  // hipMallocManaged(&y, N*sizeof(float));

  // // initialize x and y arrays on the host
  // for (int i = 0; i < N; i++) {
  //   x[i] = 1.0f;
  //   y[i] = 2.0f;
  // }

  // // Run kernel on 1M elements on the GPU
  // add<<<1, 1>>>(N, x, y);

  // // Wait for GPU to finish before accessing on host
  // hipDeviceSynchronize();

  // // Check for errors (all values should be 3.0f)
  // float maxError = 0.0f;
  // for (int i = 0; i < N; i++)
  //   maxError = fmax(maxError, fabs(y[i]-3.0f));
  // std::cout << "Max error: " << maxError << std::endl;

  // // Free memory
  // hipFree(x);
  // hipFree(y);
  
  return 0;
}


